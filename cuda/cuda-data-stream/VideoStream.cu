#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <stdlib.h>
#include <stdio.h>
#include <string>

#include "../rf-time.h"

using namespace std;

#ifdef __cplusplus
 #define __STDC_CONSTANT_MACROS
 #ifdef _STDINT_H
  #undef _STDINT_H
 #endif
 # include <stdint.h>
#endif

//#define SDL_INTERFACE

#define CUDA_SAFE_CALL
#define ELEM(i,j,DIMX_) ((i)+(j)*(DIMX_))

#define BUS_SZ 8

hipEvent_t     start, stop;
hipStream_t    stream0, stream1;
float           elapsedTime, totalTime;

extern "C" {

#include <libavcodec/avcodec.h>
#include <libavformat/avformat.h>
#include <libswscale/swscale.h>
#include <libavutil/pixfmt.h>
#include <libavutil/mathematics.h>

#include <SDL.h>
#include <SDL_thread.h>

AVFormatContext * pFormatCtx;
AVCodecContext * pCodecCtx, * c = NULL;
AVCodec * pCodec;
AVFrame * pFrameRGB, * pDecodedFrame, * pOutputFrame;
AVPacket packet;
struct SwsContext * sws_ctx = NULL, * out_sws_ctx = NULL;

int video_stream = -1;

SDL_Overlay * bmp;
SDL_Rect rect;
SDL_Event event;

const char * filename;
uint8_t * bufferRGB, * bufferYUV;
uint8_t * outbuf;
int numBytesRGB, numBytesYUV;
int frameFinished;
int counter_frames = 0;

int outbuf_size = 300000, out_size;

FILE * pFile;

string filter_type;

int blSizeX = 16, blSizeY = 16;
int numBlur = 1;
int isComplex = 1;

unsigned char * d_image1 = NULL;
unsigned char * d_image2 = NULL;
unsigned char * d_image3 = NULL;
unsigned char * d_image4 = NULL;
unsigned char * d_image5 = NULL;
unsigned char * d_image6 = NULL;
unsigned char * d_image7 = NULL;
unsigned char * d_image8 = NULL;

ofstream logfile;

__host__ int setup_video(const char * filename);
__host__ SDL_Overlay * init_sdl_window(AVCodecContext * pCodecCtx, SDL_Overlay * bmp);
__host__ void play_original_video(const char * arg);
__host__ void filter_video(AVFrame * pFrame, int width, int height);
__host__ void cuda_init(int h_height, int h_width);
__host__ void cuda_finish();
__global__ void grayGPU(unsigned char *image, int width, int height);
__global__ void blurGPUSimplex(unsigned char * image, int width, int height); 
__global__ void blurGPUComplex(unsigned char * image, int width, int height); 


__host__ int main (int argc, char ** argv)
{
	if( argc == 1 )
	{
		fprintf(stderr, "Para rodar o programa, use: %s  <video> [BlockDimX] [BlockDimY] <filter-type>\n", argv[0]);
		return -1;
	}
	switch( argc ) {

	case 3:
		blSizeX = atoi( argv[2] );
		break;
	case 4:
		blSizeX = atoi( argv[2] );
		blSizeY = atoi( argv[3] );
		break;
	case 5:
		blSizeX = atoi( argv[2] );
		blSizeY = atoi( argv[3] );
		numBlur = atoi( argv[4] );
		break;
	case 6: 
		blSizeX = atoi( argv[2] );
		blSizeY = atoi( argv[3] );
		numBlur = atoi( argv[4] );
		filter_type = string(argv[5]);
		break;
	}

	fprintf( stderr, "Tamanho do Bloco (%d,%d): %d\n", blSizeX, blSizeY, blSizeX*blSizeY );
	fprintf( stderr, "Numero de filtragens: %d\n", numBlur );
	fprintf(stderr, "Tipo do filtro: %s\n", filter_type.c_str() );

	if (filter_type == "simplex")
	{
		isComplex = 0;
	}
	else if (filter_type == "complex")
	{
		isComplex = 1;
	}
	else
	{
		fprintf(stderr, "Filtro não existe \n");
		return 1;
	}

	if (setup_video(argv[1]) < 0)
		return -1;

	cuda_init(pCodecCtx->width, pCodecCtx->height);
	double start_time;
	while(av_read_frame(pFormatCtx, &packet) >= 0)
	{
	  	//Testa se e unm pacote com de stream de video
	  	if(packet.stream_index == video_stream) 
	  	{	  		
	  		start_time = get_clock_msec();
			// Decode frame de video
		    avcodec_decode_video2(pCodecCtx, pDecodedFrame, &frameFinished, &packet);
		    
		    //Testa se ja existe um quadro de video
		    if (frameFinished) 
		    {
		    
		    #ifdef SDL_INTERFACE	
		    	SDL_LockYUVOverlay(bmp);
		    #endif

		   		//Converte a imagem de seu formato nativo para RGB
		   		sws_scale
				(
					sws_ctx,
					(uint8_t const * const *) pDecodedFrame->data,
					pDecodedFrame->linesize,
					0,
					pCodecCtx->height,
					pFrameRGB->data,
					pFrameRGB->linesize
				);

				filter_video(pFrameRGB, pCodecCtx->width, pCodecCtx->height);

			#if defined(SDL_INTERFACE) || defined(SAVE_VIDEO)
				//Convertendo de RGB para YUV
				sws_scale (
					out_sws_ctx, 
					(uint8_t const * const *) pFrameRGB->data, 
					pFrameRGB->linesize, 
        			0, 
        			c->height, 
        			pOutputFrame->data, 
        			pOutputFrame->linesize
        		);	
			#endif

			#ifdef SDL_INTERFACE	
				
	    		pOutputFrame->data[0] = bmp->pixels[0];
				pOutputFrame->data[1] = bmp->pixels[2];
				pOutputFrame->data[2] = bmp->pixels[1];

				pOutputFrame->linesize[0] = bmp->pitches[0];
				pOutputFrame->linesize[1] = bmp->pitches[2];
				pOutputFrame->linesize[2] = bmp->pitches[1];

				SDL_UnlockYUVOverlay(bmp);

				rect.x = 0;
				rect.y = 0;
				rect.w = 1280;
				rect.h = 720;

				SDL_DisplayYUVOverlay(bmp, &rect);

			#endif

			#ifdef SAVE_VIDEO
				//codigo para salvar frames em uma saida
				fflush(stdout);
				out_size = avcodec_encode_video(c, outbuf, outbuf_size, pOutputFrame);
				std::cout << "write frame " << counter_frames << "(size = " << out_size << ")" << std::endl;
				fwrite(outbuf, 1, out_size, pFile);
			#endif
				// cout << "Frame [" << counter_frames <<"] : " << get_clock_msec() - start_time<< " ms" << endl;
				// counter_frames++;

		     }	
    	}
    
  		// Libera o pacote alocado pelo pacote
  		av_free_packet(&packet);
  	
  	#ifdef SDL_INTERFACE	
  		SDL_PollEvent(&event);
	    
	    switch(event.type) 
	    {
	    	case SDL_QUIT: SDL_Quit();
	    		return 0;
	      		break;
	    	default:
	      		break;
	    }
	#endif
	}

#ifdef SAVE_VIDEO	
	//captura frames atrasados 
    for(; out_size; counter_frames++) { 
        fflush(stdout); 
                
        out_size = avcodec_encode_video(c, outbuf, outbuf_size, NULL); 
		std::cout << "write frame " << counter_frames << "(size = " << out_size << ")" << std::endl;
        fwrite(outbuf, 1, outbuf_size, pFile);       
    } 

	// adiciona sequencia para um real mpeg
    outbuf[0] = 0x00;
    outbuf[1] = 0x00;
    outbuf[2] = 0x01;
    outbuf[3] = 0xb7;
#endif

#ifdef SAVE_VIDEO
	fwrite(outbuf, 1, 4, pFile);
	fclose(pFile);
#endif

	logfile << "total time: " << totalTime << endl;

	free(outbuf);

	cuda_finish();

	av_free(bufferRGB);
	av_free(bufferYUV);

	//Fecha o codec
	avcodec_close(pCodecCtx);
	avcodec_close(c);

	//Fecha o arquivo de video
	avformat_close_input(&pFormatCtx);

	return 0;
}

__host__ int setup_video(const char * filename) 
{
	//Registra todos os codecs e formatos de videos
	av_register_all();

	//Abre o arquivo de midia;	
	if (avformat_open_input(&pFormatCtx, filename, NULL, NULL)!=0) 
	{
		fprintf(stderr, "Nao foi possivel abrir o arquivo %s\n", filename);
		return -1;
	}

	//Recupera a informacao do stream;
	if (avformat_find_stream_info(pFormatCtx, NULL) < 0)
  	{
  		fprintf(stderr, "Nao foi possivel encontrar a informacao do stream\n");
  		return -1; 
  	}

  	//Informacao bruta sobre o arquivo de video;
	av_dump_format(pFormatCtx, 0, filename, 0);

	//Encontra o primeiro stream de video (video principal)	
	for (unsigned i = 0; i < pFormatCtx->nb_streams; i++)
	{
		if(pFormatCtx->streams[i]->codec->codec_type == AVMEDIA_TYPE_VIDEO)
		{
			video_stream = i;
			break;
		}
	}

	if (video_stream == -1)
	{
		fprintf(stderr, "Nao foi possivel encontrar o stream de video\n");
		return -1;
	}

	//Captura o ponteiro referente ao codec do stream de video
	pCodecCtx = pFormatCtx->streams[video_stream]->codec;

	//Busca o decode do video
	if ((pCodec = avcodec_find_decoder(pCodecCtx->codec_id)) == NULL)
	{
		fprintf(stderr, "Codec nao suportado :(\n");
		return -1;
	}

	//Abre o codec	
	if (avcodec_open2(pCodecCtx, pCodec, NULL) < 0)
	{
		fprintf(stderr, "Nao foi possivel abrir o codec\n");
	}

	// Aloca espaco de memoria para o frame de video (AVFrame)
	pDecodedFrame = avcodec_alloc_frame();

	if ((pFrameRGB = avcodec_alloc_frame()) == NULL)
  	{
  		fprintf(stderr, "Nao foi possivel alocar memoria para o frame de video\n");
	  	return -1;
  	}

  	if ((pOutputFrame = avcodec_alloc_frame()) == NULL)
  	{
  		fprintf(stderr, "Nao foi possivel alocar memoria para o frame de video\n");
	  	return -1;
  	}
	
	//Determina o tamanho necessario do buffer e aloca a memoria
	numBytesRGB = avpicture_get_size(PIX_FMT_RGB24, pCodecCtx->width, pCodecCtx->height);
	
	bufferRGB = (uint8_t *) av_malloc(numBytesRGB*sizeof(uint8_t));

	//Configura o contexto para o escalonamento
	sws_ctx = sws_getContext (
	        pCodecCtx->width,
	        pCodecCtx->height,
	        pCodecCtx->pix_fmt,
	        pCodecCtx->width,
	        pCodecCtx->height,
	        PIX_FMT_RGB24,
	        SWS_BILINEAR,
	        NULL,
	        NULL,
	        NULL
	);

	//Aplica para o buffer os frames no formato FMT_RGB24 (pacote RGB 8:8:8, 24bpp, RGBRGB...)
	avpicture_fill((AVPicture *) pFrameRGB, bufferRGB , PIX_FMT_RGB24, pCodecCtx->width, pCodecCtx->height);

	//Preparando AVCodecContext de saida
	AVCodec * codec = avcodec_find_encoder(CODEC_ID_MPEG1VIDEO);

	if (!codec)
	{
		fprintf(stderr, "Codec nao encontrado\n");
		return -1;
	}

	c = avcodec_alloc_context3(codec);

	//Configurando valores para o contexto do video de saida
    c->bit_rate = pCodecCtx->bit_rate;
    c->width = pCodecCtx->width;
    c->height = pCodecCtx->height;
    c->time_base = pCodecCtx->time_base;
    c->gop_size = pCodecCtx->gop_size;
    c->max_b_frames = pCodecCtx->max_b_frames;
    c->pix_fmt = PIX_FMT_YUV420P;

    if (avcodec_open2(c, codec, NULL) < 0) return -1;

#ifdef SAVE_VIDEO
    pFile = fopen("out.mpg", "wb");
	if (!pFile) 
	{
    	fprintf(stderr, "could not open out.mpg\n");
	    return -1;
	}
#endif
	outbuf = (uint8_t *) av_malloc(outbuf_size);


	//Criacao de contexto para converter um tipo RGB24 para YUV240P (preparacao para encoded)
    numBytesYUV = avpicture_get_size(PIX_FMT_YUV420P, c->width, c->height);
	
	bufferYUV = (uint8_t *) av_malloc(numBytesYUV*sizeof(uint8_t));

    out_sws_ctx = sws_getContext (
	        c->width,
	        c->height,
	        PIX_FMT_RGB24,
	       	c->width,
	        c->height,
	        PIX_FMT_YUV420P,
	        SWS_FAST_BILINEAR,
	        NULL,
	        NULL,
	        NULL
	);

	avpicture_fill((AVPicture *) pOutputFrame, bufferYUV , PIX_FMT_YUV420P, c->width, c->height);


#ifdef SDL_INTERFACE	
	bmp = init_sdl_window(pCodecCtx, bmp);
	
	if (bmp == NULL) 
	{
		return -1;
	}
	
	// play_original_video(filename);
#endif

}

__host__ SDL_Overlay * init_sdl_window(AVCodecContext * pCodecCtx, SDL_Overlay * bmp) 
{
	if (SDL_Init(SDL_INIT_VIDEO | SDL_INIT_AUDIO | SDL_INIT_TIMER)) 
	{
    	fprintf(stderr, "Nao foi possivel inicializar o SDL - %s\n", SDL_GetError());
    	return NULL;
  	}

  	SDL_Surface * screen;

	screen = SDL_SetVideoMode(1280, 720, 0, 0);
	if (!screen) 
	{
  		fprintf(stderr, "SDL: Nao foi possivel configurar o modo do video\n");
  		return NULL;
	}

	bmp = SDL_CreateYUVOverlay(pCodecCtx->width, pCodecCtx->height, SDL_YV12_OVERLAY, screen);
	
  	return bmp;
}

__host__ void play_original_video(const char * arg) 
{
	char command[50];
	sprintf(command, "vlc %s &",arg);
	system(command);
}

__host__ void cuda_init(int h_width, int h_height)
{
	int  size = 3 * h_height * h_width;
	CUDA_SAFE_CALL(hipHostAlloc((void**) &pFrameRGB->data[0], size, hipHostMallocDefault));
	CUDA_SAFE_CALL(hipMalloc((void**) &d_image1, size / BUS_SZ));
	CUDA_SAFE_CALL(hipMalloc((void**) &d_image2, size / BUS_SZ));
	CUDA_SAFE_CALL(hipMalloc((void**) &d_image3, size / BUS_SZ));
	CUDA_SAFE_CALL(hipMalloc((void**) &d_image4, size / BUS_SZ));
	CUDA_SAFE_CALL(hipMalloc((void**) &d_image5, size / BUS_SZ));
	CUDA_SAFE_CALL(hipMalloc((void**) &d_image6, size / BUS_SZ));
	CUDA_SAFE_CALL(hipMalloc((void**) &d_image7, size / BUS_SZ));
	CUDA_SAFE_CALL(hipMalloc((void**) &d_image8, size / BUS_SZ));

	CUDA_SAFE_CALL(hipStreamCreate(&stream0));
	CUDA_SAFE_CALL(hipStreamCreate(&stream1));

	CUDA_SAFE_CALL(hipEventCreate(&start));
	CUDA_SAFE_CALL(hipEventCreate(&stop));
	logfile.open("log.txt", ofstream::out | ofstream::app);
}

__host__ void cuda_finish() 
{
	CUDA_SAFE_CALL(hipHostFree(pFrameRGB->data[0]));
	CUDA_SAFE_CALL(hipFree(d_image1));
	CUDA_SAFE_CALL(hipFree(d_image2));
	CUDA_SAFE_CALL(hipFree(d_image3));
	CUDA_SAFE_CALL(hipFree(d_image4));
	CUDA_SAFE_CALL(hipFree(d_image5));
	CUDA_SAFE_CALL(hipFree(d_image6));
	CUDA_SAFE_CALL(hipFree(d_image7));
	CUDA_SAFE_CALL(hipFree(d_image8));

	CUDA_SAFE_CALL(hipStreamDestroy(stream0));
	CUDA_SAFE_CALL(hipStreamDestroy(stream1));
}

__host__ void filter_video(AVFrame * pFrame, int h_width, int h_height)
{
	int  size = 3 * h_height * h_width;
	int N = size / BUS_SZ;

	// Calcula dimensoes da grid e dos blocos
	dim3 blockSize( blSizeX, blSizeY );
	int numBlocosX = h_width  / blockSize.x + ( h_width  % blockSize.x == 0 ? 0 : 1 );
	int numBlocosY = h_height / blockSize.y + ( h_height % blockSize.y == 0 ? 0 : 1 );
	dim3 gridSize( numBlocosX, numBlocosY / BUS_SZ, 1 );

	hipEventRecord(start, 0);

	CUDA_SAFE_CALL(hipMemcpyAsync(d_image1, pFrameRGB->data[0], N, hipMemcpyHostToDevice, stream0));
	CUDA_SAFE_CALL(hipMemcpyAsync(d_image2, pFrameRGB->data[0] + N, N, hipMemcpyHostToDevice, stream1));
	
	// grayGPU<<< gridSize, blockSize, 0, stream0 >>>(d_image1, h_width, 1 + h_height / BUS_SZ);
	// grayGPU<<< gridSize, blockSize, 0, stream1 >>>(d_image2, h_width, h_height / BUS_SZ);

	for (int i = 0; i < numBlur; i++)
	{
		if (isComplex)
		{
			blurGPUComplex<<< gridSize, blockSize, 0, stream0 >>>(d_image1, h_width, h_height / BUS_SZ);	
			blurGPUComplex<<< gridSize, blockSize, 0, stream1 >>>(d_image2, h_width, h_height / BUS_SZ);		
		}
		else
		{
			blurGPUSimplex<<< gridSize, blockSize, 0, stream0 >>>(d_image1, h_width, h_height / BUS_SZ);	
			blurGPUSimplex<<< gridSize, blockSize, 0, stream1 >>>(d_image2, h_width, h_height / BUS_SZ);			
		}
		
	}

	CUDA_SAFE_CALL(hipMemcpyAsync(pFrameRGB->data[0], d_image1, N, hipMemcpyDeviceToHost, stream0));
	CUDA_SAFE_CALL(hipMemcpyAsync(pFrameRGB->data[0] + N, d_image2, N, hipMemcpyDeviceToHost, stream1));

	CUDA_SAFE_CALL(hipMemcpyAsync(d_image3, pFrameRGB->data[0] + 2 * N, N, hipMemcpyHostToDevice, stream0));
	CUDA_SAFE_CALL(hipMemcpyAsync(d_image4, pFrameRGB->data[0] + 3 * N, N, hipMemcpyHostToDevice, stream1));

	// grayGPU<<< gridSize, blockSize, 0, stream0 >>>(d_image3, h_width, h_height / BUS_SZ);
	// grayGPU<<< gridSize, blockSize, 0, stream1 >>>(d_image4, h_width, h_height / BUS_SZ);

	for (int i = 0; i < numBlur; i++)
	{
		if (isComplex)
		{
			blurGPUComplex<<< gridSize, blockSize, 0, stream0 >>>(d_image1, h_width, h_height / BUS_SZ);	
			blurGPUComplex<<< gridSize, blockSize, 0, stream1 >>>(d_image2, h_width, h_height / BUS_SZ);		
		}
		else
		{
			blurGPUSimplex<<< gridSize, blockSize, 0, stream0 >>>(d_image1, h_width, h_height / BUS_SZ);	
			blurGPUSimplex<<< gridSize, blockSize, 0, stream1 >>>(d_image2, h_width, h_height / BUS_SZ);			
		}
	}

	CUDA_SAFE_CALL(hipMemcpyAsync(pFrameRGB->data[0] + 2 * N, d_image3, N, hipMemcpyDeviceToHost, stream0));
	CUDA_SAFE_CALL(hipMemcpyAsync(pFrameRGB->data[0] + 3 * N, d_image4, N, hipMemcpyDeviceToHost, stream1));

	CUDA_SAFE_CALL(hipMemcpyAsync(d_image5, pFrameRGB->data[0] + 4 * N, N, hipMemcpyHostToDevice, stream0));
	CUDA_SAFE_CALL(hipMemcpyAsync(d_image6, pFrameRGB->data[0] + 5 * N, N, hipMemcpyHostToDevice, stream1));

	// grayGPU<<< gridSize, blockSize, 0, stream0 >>>(d_image5, h_width, 3 + h_height / BUS_SZ);
	// grayGPU<<< gridSize, blockSize, 0, stream1 >>>(d_image6, h_width, 3 + h_height / BUS_SZ);

	for (int i = 0; i < numBlur; i++)
	{
		if (isComplex)
		{
			blurGPUComplex<<< gridSize, blockSize, 0, stream0 >>>(d_image1, h_width, h_height / BUS_SZ);	
			blurGPUComplex<<< gridSize, blockSize, 0, stream1 >>>(d_image2, h_width, h_height / BUS_SZ);		
		}
		else
		{
			blurGPUSimplex<<< gridSize, blockSize, 0, stream0 >>>(d_image1, h_width, h_height / BUS_SZ);	
			blurGPUSimplex<<< gridSize, blockSize, 0, stream1 >>>(d_image2, h_width, h_height / BUS_SZ);			
		}
	}

	CUDA_SAFE_CALL(hipMemcpyAsync(pFrameRGB->data[0] + 4 * N, d_image5, N, hipMemcpyDeviceToHost, stream0));
	CUDA_SAFE_CALL(hipMemcpyAsync(pFrameRGB->data[0] + 5 * N, d_image6, N, hipMemcpyDeviceToHost, stream1));

	CUDA_SAFE_CALL(hipMemcpyAsync(d_image7, pFrameRGB->data[0] + 6 * N, N, hipMemcpyHostToDevice, stream0));
	CUDA_SAFE_CALL(hipMemcpyAsync(d_image8, pFrameRGB->data[0] + 7 * N, N, hipMemcpyHostToDevice, stream1));

	// grayGPU<<< gridSize, blockSize, 0, stream0 >>>(d_image7, h_width, h_height / BUS_SZ);
	// grayGPU<<< gridSize, blockSize, 0, stream1 >>>(d_image8, h_width, h_height / BUS_SZ);

	for (int i = 0; i < numBlur; i++)
	{
		if (isComplex)
		{
			blurGPUComplex<<< gridSize, blockSize, 0, stream0 >>>(d_image1, h_width, h_height / BUS_SZ);	
			blurGPUComplex<<< gridSize, blockSize, 0, stream1 >>>(d_image2, h_width, h_height / BUS_SZ);		
		}
		else
		{
			blurGPUSimplex<<< gridSize, blockSize, 0, stream0 >>>(d_image1, h_width, h_height / BUS_SZ);	
			blurGPUSimplex<<< gridSize, blockSize, 0, stream1 >>>(d_image2, h_width, h_height / BUS_SZ);			
		}
	}

	CUDA_SAFE_CALL(hipMemcpyAsync(pFrameRGB->data[0] + 6 * N, d_image7, N, hipMemcpyDeviceToHost, stream0));
	CUDA_SAFE_CALL(hipMemcpyAsync(pFrameRGB->data[0] + 7 * N, d_image8, N, hipMemcpyDeviceToHost, stream1));


	CUDA_SAFE_CALL(hipEventRecord(stop, 0));
	CUDA_SAFE_CALL(hipEventSynchronize(stop));
	CUDA_SAFE_CALL(hipEventElapsedTime(&elapsedTime, start, stop));

	totalTime += elapsedTime;
	logfile << elapsedTime  << endl;

}

__global__ void grayGPU(unsigned char * image, int width, int height) 
{

	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;

	if(i < width && j < height) {

	 	int idx = 3*ELEM(i, j, width);
	 	int r = image[ idx+2 ];
	 	int g = image[ idx+1 ];
	 	int b = image[ idx   ];
		float gg = (float)r*0.299f +(float)g*0.587f + (float)b*0.114f;
	 	int gray = (int)gg;
	 	image[ idx   ] = (unsigned char)gray;
	 	image[ idx+1 ] = (unsigned char)gray;
	 	image[ idx+2 ] = (unsigned char)gray;		
	 }
}

__global__ void blurGPUSimplex(unsigned char * image, int width, int height)
{

	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;

	if( i > 0 && j > 0 && i < (width - 1) && j < (height - 1) )
	{

		//pixel b
		int idx = 3*ELEM( i, j, width );
		int center = image[ idx ];
		int left      = image[ idx-3 ];
		int right    = image[ idx+3 ];
		int top      = image[ idx+3*width ];
		int bottom = image[ idx-3*width ];

		float gg = (float) ((float) center + (float) left + (float)
				    right + (float) top + (float) bottom)/5.0f;
		int blur = (int)gg;
		image[ idx ] = (unsigned char)blur;

		//pixel g
		idx++;
		center = image[ idx ];
		left      = image[ idx-3 ];
		right    = image[ idx+3 ];
		top      = image[ idx+3*width ];
		bottom = image[ idx-3*width ];

		gg = (float) ((float) center + (float) left + (float)
				    right + (float) top + (float) bottom)/5.0f;
		blur = (int)gg;
		image[ idx ] = (unsigned char)blur;


		//pixel r
		idx++;
		center = image[ idx ];
		left      = image[ idx-3 ];
		right    = image[ idx+3 ];
		top      = image[ idx+3*width ];
		bottom = image[ idx-3*width ];

		gg = (float) ((float) center + (float) left + (float)
				    right + (float) top + (float) bottom)/5.0f;
		blur = (int)gg;
		image[ idx ] = (unsigned char)blur;
	}

}

__global__ void blurGPUComplex(unsigned char * image, int width, int height)
{

	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;

	if( i > 0 && j > 0 && i < (width - 1) && j < (height - 1) )
	{

		//pixel b
		int idx = 3*ELEM( i, j, width );
		int center = image[ idx ];
		int left      = image[ idx-3 ];
		int right    = image[ idx+3 ];
		int top      = image[ idx+3*width ];
		int bottom = image[ idx-3*width ];

		float k1 = sqrt((float)((center - left)*(center - left)));
		float k2 = sqrt((float)((center - right)*(center - right)));
		float k3 = sqrt((float)((center - top)*(center - top)));
		float k4 = sqrt((float)((center - bottom)*(center - bottom)));

		float gg = (float) ((float) center + (float) left*k1 + (float)
				    right*k2 + (float) top*k3 + (float) bottom*k4)/(1 + k1 + k2 + k3 +
										    k4);
		int blur = (int)gg;
		image[ idx ] = (unsigned char)blur;

		//pixel g
		idx++;
		center = image[ idx ];
		left      = image[ idx-3 ];
		right    = image[ idx+3 ];
		top      = image[ idx+3*width ];
		bottom = image[ idx-3*width ];

		k1 = sqrt((float)((center - left)*(center - left)));
		k2 = sqrt((float)((center - right)*(center - right)));
		k3 = sqrt((float)((center - top)*(center - top)));
		k4 = sqrt((float)((center - bottom)*(center - bottom)));

		gg = (float) ((float) center + (float) left*k1 + (float)
			      right*k2 + (float) top*k3 + (float) bottom*k4)/(1 + k1 + k2 + k3 +
									      k4);
		blur = (int)gg;
		image[ idx ] = (unsigned char)blur;


		//pixel r
		idx++;
		center = image[ idx ];
		left      = image[ idx-3 ];
		right    = image[ idx+3 ];
		top      = image[ idx+3*width ];
		bottom = image[ idx-3*width ];

		k1 = sqrt((float)((center - left)*(center - left)));
		k2 = sqrt((float)((center - right)*(center - right)));
		k3 = sqrt((float)((center - top)*(center - top)));
		k4 = sqrt((float)((center - bottom)*(center - bottom)));

		gg = (float) ((float) center + (float) left*k1 + (float)
			      right*k2 + (float) top*k3 + (float) bottom*k4)/(1 + k1 + k2 + k3 +
									      k4);
		blur = (int)gg;
		image[ idx ] = (unsigned char)blur;
	}

}

}